#include "hip/hip_runtime.h"
#include "find_min_max.h"

#include <stk/cuda/hip/hip_runtime.h>
#include <stk/cuda/ptr.h>

#include <stk/image/gpu_volume.h>

#include <algorithm>
#include <cfloat>

namespace cuda = stk::cuda;


// Implementation 1
// Interleaved addressing w/o any optimization

__global__ void algo1_reduce_volume_min_max(
    const cuda::VolumePtr<float> in,
    dim3 dims,
    float2* out)
{
    extern __shared__ float2 shared[];

    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int bid = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;

    shared[tid] = {FLT_MAX, -FLT_MAX};

    if (x < dims.x &&
        y < dims.y &&
        z < dims.z) {
        shared[tid].x = in(x,y,z);
        shared[tid].y = in(x,y,z);
    }
    __syncthreads();

    for (unsigned int s=1; s < blockDim.x*blockDim.y*blockDim.z; s *= 2)
    {
        if ((tid % (2*s)) == 0)
        {
            shared[tid].x = min(shared[tid].x, shared[tid + s].x);
            shared[tid].y = max(shared[tid].y, shared[tid + s].y);
        }

        __syncthreads();
    }

    if (tid == 0) out[bid] = shared[0];
}

__global__ void algo1_reduce_min_max(
    unsigned int n,
    float2* in,
    float2* out)
{
    extern __shared__ float2 shared[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x) + threadIdx.x;

    if (i < n) shared[tid] = in[i];
    else shared[tid] = {FLT_MAX, -FLT_MAX};

    __syncthreads();

    for (unsigned int s=1; s < blockDim.x; s *= 2)
    {
        if ((tid % (2*s)) == 0)
        {
            shared[tid].x = min(shared[tid].x, shared[tid + s].x);
            shared[tid].y = max(shared[tid].y, shared[tid + s].y);
        }

        __syncthreads();
    }

    if (tid == 0) out[blockIdx.x] = shared[0];
}


void find_min_max_1(stk::GpuVolume& vol, float& min, float& max)
{
    dim3 block_size{8,8,8};

    dim3 grid_size {
        (vol.size().x + block_size.x - 1) / block_size.x,
        (vol.size().y + block_size.y - 1) / block_size.y,
        (vol.size().z + block_size.z - 1) / block_size.z
    };

    // Number of blocks (or values in the active buffer)
    uint32_t n = grid_size.x * grid_size.y * grid_size.z;

    // Allocate our global buffers
    float2* d_out;
    CUDA_CHECK_ERRORS(hipMalloc(&d_out, 2*n*sizeof(float)));

    float2* d_in;
    CUDA_CHECK_ERRORS(hipMalloc(&d_in, 2*n*sizeof(float)));

    algo1_reduce_volume_min_max<<<grid_size, block_size, 
        uint32_t(2*sizeof(float)*512)>>>(
        vol, vol.size(), d_out
    );

    CUDA_CHECK_ERRORS(hipPeekAtLastError());
    CUDA_CHECK_ERRORS(hipDeviceSynchronize());

    while (n > 1) {
        // block_count should always be pow2 as it follows the gridsize from 
        //  previous step
        uint32_t n_threads = std::min<uint32_t>(n, 1024);
        uint32_t n_blocks = (n + n_threads - 1) / n_threads;

        CUDA_CHECK_ERRORS(hipMemcpy(d_in, d_out, 2*n*sizeof(float), 
            hipMemcpyDeviceToDevice));

        algo1_reduce_min_max<<<{n_blocks,1,1}, {n_threads,1,1}, 
                                uint32_t(2*sizeof(float)*n_threads)>>>(
            n, d_in, d_out);

        CUDA_CHECK_ERRORS(hipPeekAtLastError());
        CUDA_CHECK_ERRORS(hipDeviceSynchronize());

        n = n_blocks;
    }

    float2 min_max;
    CUDA_CHECK_ERRORS(hipMemcpy(&min_max, d_out, 2*sizeof(float), hipMemcpyDeviceToHost));

    min = min_max.x;
    max = min_max.y;

    CUDA_CHECK_ERRORS(hipFree(d_in));
    CUDA_CHECK_ERRORS(hipFree(d_out));
}

